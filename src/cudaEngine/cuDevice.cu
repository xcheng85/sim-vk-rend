#include <iostream>

#include <cuDevice.h>
#include <cuPredefines.h>

namespace core::cuda
{
    // from cuda-samples repo
    inline int _ConvertSMVer2Cores(int major, int minor)
    {
       // Defines for GPU Architecture types (using the SM version to determine
       // the # of cores per SM
       typedef struct
       {
           int SM; // 0xMm (hexidecimal notation), M = SM Major version,
           // and m = SM minor version
           int Cores;
       } sSMtoCores;

       sSMtoCores nGpuArchCoresPerSM[] = {
           {0x30, 192},
           {0x32, 192},
           {0x35, 192},
           {0x37, 192},
           {0x50, 128},
           {0x52, 128},
           {0x53, 128},
           {0x60, 64},
           {0x61, 128},
           {0x62, 128},
           {0x70, 64},
           {0x72, 64},
           {0x75, 64},
           {0x80, 64},
           {0x86, 128},
           {0x87, 128},
           {0x89, 128},
           {0x90, 128},
           {-1, -1}};

       int index = 0;

       while (nGpuArchCoresPerSM[index].SM != -1)
       {
           if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
           {
               return nGpuArchCoresPerSM[index].Cores;
           }

           index++;
       }

       // If we don't find the values, we default use the previous one
       // to run properly
       printf(
           "MapSMtoCores for SM %d.%d is undefined."
           "  Default to use %d Cores/SM\n",
           major, minor, nGpuArchCoresPerSM[index - 1].Cores);
       return nGpuArchCoresPerSM[index - 1].Cores;
    }

    void selectDevice()
    {
        int nDevices;
        hipGetDeviceCount(&nDevices);
        printf("Number of devices: %d\n", nDevices);
        size_t maxGFLOPS = 0;
        int selectedDeviceId = 0;
        for (int i = 0; i < nDevices; i++)
        {
           hipDeviceProp_t prop;
           hipGetDeviceProperties(&prop, i);
           printf("Device Number: %d\n", i);
           printf("  Device name: %s\n", prop.name);
           printf("  Memory Clock Rate (MHz): %d\n",
                  prop.memoryClockRate / 1024);
           printf("  Memory Bus Width (bits): %d\n",
                  prop.memoryBusWidth);
           printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
                  2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);

           printf("  Total global memory (Gbytes) %.1f\n", (float)(prop.totalGlobalMem) / 1024.0 / 1024.0 / 1024.0);
           printf("  Total constant memory (Kbytes) %.1f\n", (float)(prop.totalConstMem) / 1024.0);
           printf("  Shared memory per block (Kbytes) %.1f\n", (float)(prop.sharedMemPerBlock) / 1024.0);
           printf("  Max Threads per block %.1f\n", (float)(prop.maxThreadsPerBlock));
           printf("  Max Threads per SM %.1f\n", (float)(prop.maxThreadsPerMultiProcessor));
           printf("  Max Registers per block %.1f\n", (float)(prop.regsPerBlock));
           printf("  Max Registers per SM %.1f\n", (float)(prop.regsPerMultiprocessor));
           printf("  SM count: %d\n", prop.multiProcessorCount);
           printf("  minor-major: %d-%d\n", prop.minor, prop.major);
           printf("  Warp-size: %d\n", prop.warpSize);
           printf("  device overlap (perform hipMemcpy and kernel exec simultaneously): %s\n", prop.deviceOverlap ? "yes" : "no");
           printf("  canMapHostMemoryTo Device Address space: %s\n", prop.canMapHostMemory ? "yes" : "no");
           printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
           printf("  Concurrent computation/communication: %s\n\n", prop.deviceOverlap ? "yes" : "no");

           // hipError_t
           int computeMode = -1, multiProcessorCount = 0, clockRate = 0;
           CUDA_CHECK(hipDeviceGetAttribute(&computeMode, hipDeviceAttributeComputeMode, i));
           CUDA_CHECK(hipDeviceGetAttribute(&clockRate, hipDeviceAttributeClockRate, i));
           CUDA_CHECK(hipDeviceGetAttribute(&multiProcessorCount, hipDeviceAttributeMultiprocessorCount, i));
           auto numSmPerMultiProcessor = _ConvertSMVer2Cores(prop.major, prop.minor);
           auto GFLOPS = multiProcessorCount * numSmPerMultiProcessor * clockRate;
           // GFLOPS stands for "Giga Floating Point Operations Per Second." It is a measure of a co
           std::cout << "GFLOPS: " << GFLOPS << std::endl;

           if (GFLOPS > maxGFLOPS)
           {
               maxGFLOPS = GFLOPS;
               selectedDeviceId = i;
           }
        }
        // since cuda12.0, hipSetDevice will now explicitly initialize the runtime. check for initialize errors.
        CUDA_CHECK(hipSetDevice(selectedDeviceId));
    }
}